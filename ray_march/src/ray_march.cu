#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "cutil_math.h"

#define NUM_THREADS 1024
#define NUM_BLOCKS 256
#define FOCAL_LENGTH 1095
#define MAX_HIT 8


__device__ __constant__ float c_center[3]; // camera center
__device__ __constant__ float c_rotation[3][3]; // camera rotation

__device__ __forceinline__ int sign(float x) { 
	int t = x<0 ? -1 : 0;
	return x > 0 ? 1 : t;
}

/**
 * Find the cloest hit on the voxel grid given rays
 * @param batch_size number of pixels
 * @param o intersection location buffer: hit x h x w x 6
 * @param v ray direction buffer
 * @param finish mask indicate whether the evaluation of a pixel is finished
 * @param rgba image rgba buffer
 * @param octrees flattened dense octrees of occupancy mask
 */
template<int voxel_num>
__global__ void aabb_intersect_kernel(
            const int batch_size,
            float *__restrict__ o,
            float *__restrict__ v,
            bool* __restrict__ finish,
            float* __restrict__ rgba,
            const bool* __restrict__ octrees) {

    int batch_index = (blockIdx.x * blockDim.x) + threadIdx.x; 


    if(batch_index >= batch_size){
        return;
    }

    // initialize buffer
    o += batch_size*6*(MAX_HIT-1) + batch_index*6 + 3;
    v += batch_index*3;
    finish += batch_index;
    rgba += batch_index*4;
    rgba[0] = 0.0f;
    rgba[1] = 0.0f;
    rgba[2] = 0.0f;
    rgba[3] = 1.0f;
    finish[0] = false;


    // get current viewing direction
    float v_ori[3];
    v_ori[0] = (399.5f-(batch_index%800))/FOCAL_LENGTH;
    v_ori[1] = (399.5f - batch_index/800)/FOCAL_LENGTH;
    v_ori[2] = -1.0f;
    float v_new[3];

    #pragma unroll
    for (int i = 0; i < 3;i++) {
        v_new[i] = v_ori[0]*c_rotation[i][0]+v_ori[1]*c_rotation[i][1]+v_ori[2]*c_rotation[i][2];
    }
    float norm = sqrt(v_new[0]*v_new[0]+v_new[1]*v_new[1]+v_new[2]*v_new[2]);
    v_new[0]/=norm;
    v_new[1]/=norm;
    v_new[2]/=norm;

    float ox = c_center[0];
    float oy = c_center[1];
    float oz = c_center[2];


    float3 dir = make_float3(v_new[0],v_new[1],v_new[2]);
    float3 ori = make_float3(ox,oy,oz);

    bool is_inside = (ori.x >= 0) & (ori.y >= 0) & (ori.z >= 0) 
                & (ori.x <= voxel_num) & (ori.y <= voxel_num) & (ori.z <= voxel_num);

    if (is_inside==false) { // ray bounding volume intersection
        float t0 = (-ori.x)/dir.x;
        float t1 = (voxel_num-ori.x)/dir.x;
        float tmin = fminf(t0,t1);
        float tmax = fmaxf(t0,t1);

        t0 = (-ori.y)/dir.y;
        t1 = (voxel_num-ori.y)/dir.y;
        tmin = fmaxf(tmin, fminf(t0,t1));
        tmax = fminf(tmax, fmaxf(t0,t1));

        t0 = (-ori.z)/dir.z;
        t1 = (voxel_num-ori.z)/dir.z;
        tmin = fmaxf(tmin, fminf(t0,t1));
        tmax = fminf(tmax, fmaxf(t0,t1));

        ori.x = clamp(ori.x+dir.x*tmin, 0.0f, float(voxel_num));
        ori.y = clamp(ori.y+dir.y*tmin, 0.0f, float(voxel_num));
        ori.z = clamp(ori.z+dir.z*tmin, 0.0f, float(voxel_num));

        if (tmin > tmax) { // a miss, exit
            o[0] = -1.0f;
            finish[0] = true;
            return;
        }
    }

    ori /= 64.0f;
    float3 ori_last;
    float3 step = make_float3(sign(dir.x), sign(dir.y), sign(dir.z));
    float3 bound;
    float tx;
    float ty;
    float tz;
    int mask_size = voxel_num/64;
    int octree_offset = 0;

    while (mask_size < voxel_num) { // octree traversal
        ori_last.x = ori.x;
        ori_last.y = ori.y;
        ori_last.z = ori.z;

        bound = floor(ori_last*step+1.0f)*step;
        tx = (bound.x-ori_last.x) / dir.x;
        ty = (bound.y-ori_last.y) / dir.y;
        tz = (bound.z-ori_last.z) / dir.z;

        float tnext = fminf(tx, fminf(ty,tz));
        ori = ori_last + (dir*tnext);

        // check which planes has been hitted
        if (tnext == tx) {
            ori.x = bound.x;
        } else if (tnext ==ty) {
            ori.y = bound.y;
        } else {
            ori.z = bound.z;
        }

        // check whether go out of boundary
        if (ori.x <0 | ori.y <0 | ori.z < 0 |
            ori.x > mask_size | ori.y > mask_size | ori.z > mask_size) {
            o[0] = -1.0f;
            finish[0] = true;
            return;
        }

        // check if hit an empty space
        float3 corner = fminf(ori_last+1e-4f,ori+1e-4f);
        int corner_index = int(corner.z)*mask_size*mask_size
                         + int(corner.y)*mask_size 
                         + int(corner.x);

        if (octrees[octree_offset+corner_index]) {
            octree_offset += mask_size*mask_size*mask_size;
            ori_last *= 4.0f;
            ori.x = ori_last.x;
            ori.y = ori_last.y;
            ori.z = ori_last.z;
            mask_size *= 4;
        }
    }
    o[0] = ori_last.x;
    o[1] = ori_last.y;
    o[2] = ori_last.z;
    v[0] = dir.x;
    v[1] = dir.y;
    v[2] = dir.z;
}

/**
 * ray marching
 * @param batch_size number of pixels
 * @param o intersection location buffer: hit x h x w x 6
 * @param v ray direction buffer
 * @param mask occupancy mask
 * @param finish inicator of whether the evaluation of a ray is finished
 */
template<int voxel_num>
__global__ void ray_march_kernel(
    int batch_size,
    float* __restrict__ o,
    const float* __restrict__ v,
    const bool* __restrict__ mask,
    bool* __restrict__ finish
) {
    int batch_index = (blockIdx.x * blockDim.x) + threadIdx.x; 
    
    while (batch_index < batch_size) {
        if (finish[batch_index]) {
            batch_index += NUM_THREADS*NUM_BLOCKS;
            continue;
        }

        int last_idx = (MAX_HIT-1)*6*batch_size+batch_index*6+3;
        float3 ori_last = make_float3(o[last_idx],o[last_idx+1],o[last_idx+2]);

        o[batch_index*6] = -1.0f; // get last intersection location
        if (ori_last.x < 0.0f) {
            break;
        }
        
        float3 dir = make_float3(v[batch_index*3],v[batch_index*3+1],v[batch_index*3+2]);
        float3 step = make_float3(sign(dir.x), sign(dir.y), sign(dir.z));
        float3 bound;
        float3 ori;
        float tx;
        float ty;
        float tz;

        int hit_num = 0;
        while (hit_num < MAX_HIT) {
            bound = floor(ori_last*step+1.0f)*step;
            tx = (bound.x-ori_last.x) / dir.x;
            ty = (bound.y-ori_last.y) / dir.y;
            tz = (bound.z-ori_last.z) / dir.z;

            float tnext = fminf(tx, fminf(ty,tz));
            ori = ori_last + (dir*tnext);

            // check which planes has been hitted
            if (tnext == tx) {
                ori.x = bound.x;
            } else if (tnext ==ty) {
                ori.y = bound.y;
            } else {
                ori.z = bound.z;
            }

            // check whether go out of volume
            if (ori.x <0 | ori.y <0 | ori.z < 0 |
                ori.x >= voxel_num | ori.y >= voxel_num | ori.z >= voxel_num) {
                o[batch_index*6+hit_num*6*batch_size] = -1.0f;
                break;
            }


            // check if hit an empty space
            float3 corner = fminf(ori_last+1e-4f,ori+1e-4f);
            int corner_index = int(corner.z)*voxel_num*voxel_num+int(corner.y)*voxel_num+int(corner.x);

            if (mask[corner_index]) {
                int rec_idx = batch_index*6+hit_num*6*batch_size;
                o[rec_idx] = ori_last.x;
                o[rec_idx+1] = ori_last.y;
                o[rec_idx+2] = ori_last.z;
                o[rec_idx+3] = ori.x;
                o[rec_idx+4] = ori.y;
                o[rec_idx+5] = ori.z;
                hit_num += 1;
            }
            ori_last.x = ori.x;
            ori_last.y = ori.y;
            ori_last.z = ori.z;
        }
        batch_index += NUM_BLOCKS*NUM_THREADS;
    }
}





void aabb_intersect_wrapper(
    const int batch_size, 
    float* o, float* v, 
    const float* center, const float* rotation,
    bool* finish, float* rgba, 
    const bool* octrees,
    int voxel_num
){
    hipMemcpyToSymbol(HIP_SYMBOL(c_center), center, 3*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_rotation),rotation, 9*sizeof(float),0,hipMemcpyHostToDevice);
    
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    switch (voxel_num) {
        case 256:
            aabb_intersect_kernel<256><<<ceil(batch_size*1.0 / NUM_THREADS), NUM_THREADS,0,stream>>>(
                batch_size,
                o,v, finish, rgba,
                octrees
            );
            break;
        case 128:
            aabb_intersect_kernel<128><<<ceil(batch_size*1.0 / NUM_THREADS), NUM_THREADS,0,stream>>>(
                batch_size,
                o,v, finish, rgba,
                octrees
            );
            break;
    }
    
    CUDA_CHECK_ERRORS();
    hipDeviceSynchronize();
  }

void ray_march_wrapper(
    const int batch_size, 
    float* o, const float* v,
    const bool* mask, bool* finish,
    int voxel_num
){

    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    switch (voxel_num) {
        case 256:
            ray_march_kernel<256><<<NUM_BLOCKS, NUM_THREADS,0,stream>>>(
                batch_size,
                o,v,
                mask, finish
            );
            break;
        case 128:
            ray_march_kernel<128><<<NUM_BLOCKS, NUM_THREADS,0,stream>>>(
                batch_size,
                o,v,
                mask, finish
            );
            break;
    }

    CUDA_CHECK_ERRORS();
    hipDeviceSynchronize();
}
